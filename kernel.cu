#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#define __CUDACC_RTC__
#define __HIPCC__
#include <hip/device_functions.h>
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <stdio.h>
#include <stdlib.h>
#include <algorithm>
typedef struct {
	int width;
	int height;
	float* elements;
	int step;
} Matrix;

using namespace std;

#define BLOCK_SIZE 4

__global__ void MatrixMulKernel(const Matrix, const Matrix, Matrix);

void MatrixMul(const Matrix A, const Matrix B, Matrix C) {
	Matrix d_A;
	d_A.width = d_A.step = A.width;
	d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(float);
	hipError_t err = hipMalloc(&d_A.elements, size);
	cout << "CUDA malloc A: " << hipGetErrorString(err) << endl;
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);

	Matrix d_B;
	d_B.width = d_B.step = B.width;
	d_B.height = B.height;
	size = B.width * B.height * sizeof(float);
	err = hipMalloc(&d_B.elements, size);
	cout << "CUDA malloc B: " << hipGetErrorString(err) << endl;
	hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);

	Matrix d_C;
	d_C.width = d_C.step = C.width;
	d_C.height = C.height;
	size = C.width * C.height * sizeof(float);
	err = hipMalloc(&d_C.elements, size);
	cout << "CUDA malloc C: " << hipGetErrorString(err) << endl;

	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
	MatrixMulKernel <<<dimGrid, dimBlock>>> (d_A, d_B, d_C);
	err = hipDeviceSynchronize();
	cout << "Run kernel: " << hipGetErrorString(err) << endl;

	err = hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);
	cout << "Copy C off of device: " << hipGetErrorString(err) << endl;

	hipFree(d_A.elements);
	hipFree(d_B.elements);
	hipFree(d_C.elements);
}